#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <iostream>
#include <string>
#include <random>
#include <stdexcept>
#include <math.h>

/*
make 2d matrix of various dimensions (generalize later, this is just a first attempt)
initializations with data/arrays, from file
get shape/dims
matrix norms
*/

__global__ void fill(float* data, float val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[idx] = val;
}

__global__ void diagfill(float* data, int n, float val)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    data[(idx * n) + idx] = val;
}

__global__ void matrixAdd(float* first, float* second, float* result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = first[idx] + second[idx];
}

__global__ void matrixSub(float* first, float* second, float* result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = first[idx] - second[idx];
}

// k is the number of cols of the second matrix, sry for obscurity, i just wanted compactness
__global__ void matrixDot(float* first, float* second, float* result, int n, int k)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < n; i++)
    {
        result[idx] += first[((idx / k) * n) + i] * second[(i * k) + (idx % k)]; // TODO optimize with tiling/whatever for efficiency/cache usage
    }// also i need to ensure that the result matrix si
}

__global__ void matrixScalarMult(float* mat, float scalar, float* result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = mat[idx] * scalar;
}

__global__ void matrixScalarDiv(float* mat, float scalar, float* result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = mat[idx] / scalar;
}

__global__ void matrixTranspose(float* src, float* dest, int rows, int cols)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx / cols;
    int y = idx % cols;
    dest[(y * rows) + x] = src[idx];
}

__global__ void matrixRELU(float* src, float* dest)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (src[idx] < 0)
    {
        dest[idx] = 0;
    }
    else
    {
        dest[idx] = src[idx];
    }
}

__global__ void matrixExp(float* src, float* dest)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dest[idx] = exp(src[idx]);
}

__global__ void matrixHad(float* src1, float* src2, float* dest)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dest[idx] = src1[idx] * src2[idx];
}

__global__ void gradRELU(float* grad, float* data, float* dest) // bespoke function for relu backprop, kind of hacky/ad hoc but w/e. TODO: test
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (data[idx] >= 0)
    {
        dest[idx] += grad[idx];
    }
}

__global__ void matrixScalarReciprocal(float scalar, float* data, float* dest) // TODO test
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dest[idx] = scalar / data[idx];
}

__global__ void avgToColumn(float* src, float* dest, int cols) // TODO test, also add parallel sum reductions maybe
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < cols; i++)
    {
        dest[idx] += src[idx*cols + i];
    }
    dest[idx] /= cols;
}

class Matrix
{
public:
    int rows, cols;
    float* data;

    enum class InitType
    {
        Identity,
        Random,
        Xavier,
        He
    };

    Matrix(int r, int c) : rows(r), cols(c), data(0) {}

    Matrix(int r, int c, float val) : rows(r), cols(c) // fill with single value
    {
        hipMalloc(&data, r * c * sizeof(float));
        fill <<<1, rows * cols>>> (data, val); // TODO make this more sophisticated
        hipDeviceSynchronize();
    }

    Matrix(int r, int c, const float* input_arr) : rows(r), cols(c)
    {
        hipMalloc(&data, r * c * sizeof(float));
        hipMemcpy(data, input_arr, r * c * sizeof(float), hipMemcpyHostToDevice);
    }

    // arg constructor
    Matrix(int r, int c, InitType type) : rows(r), cols(c)
    {
        if (r < 1 || c < 1)
        {
            throw std::invalid_argument("Matrix dimensions cannot be smaller than 1.");
        }
        if (type == Matrix::InitType::Identity) // TODO make this whole thing a switch statement
        {
            if (r == c)
            {
                hipMalloc(&data, r * c * sizeof(float));
                fill <<<1, rows * cols>>> (data, 0); // TODO make this more sophisticated
                hipDeviceSynchronize();
                diagfill <<<1, rows>>> (data, rows, 1); // this as well
                hipDeviceSynchronize();
            }
            else
            {
                throw std::invalid_argument("Matrix must be square to be an identity matrix.");
            }
        }

        else if (type == Matrix::InitType::Random)
        {
            float* hostData = new float[r * c];

            std::default_random_engine generator;
            std::normal_distribution<float> distribution(0, sqrt(.01));

            for (int i = 0; i < r * c; i++)
            {
                hostData[i] = distribution(generator);
            }

            hipMalloc(&data, r * c * sizeof(float));
            hipMemcpy(data, hostData, r * c * sizeof(float), hipMemcpyHostToDevice);

            delete[] hostData;
        }

        else if (type == Matrix::InitType::Xavier)
        {
            float* hostData = new float[r * c];

            std::default_random_engine generator;
            std::normal_distribution<float> distribution(0, sqrt(2 / float(r + c)));

            for (int i = 0; i < r * c; i++)
            {
                hostData[i] = distribution(generator);
            }

            hipMalloc(&data, r * c * sizeof(float));
            hipMemcpy(data, hostData, r * c * sizeof(float), hipMemcpyHostToDevice);

            delete[] hostData;
        }

        else if (type == Matrix::InitType::He)
        {
            float* hostData = new float[r * c];

            std::default_random_engine generator;
            std::normal_distribution<float> distribution(0, sqrt(2 / float(c)));

            for (int i = 0; i < r * c; i++)
            {
                hostData[i] = distribution(generator);
            }

            hipMalloc(&data, r * c * sizeof(float));
            hipMemcpy(data, hostData, r * c * sizeof(float), hipMemcpyHostToDevice);

            delete[] hostData;
        }
    }
    
    // destructor
    ~Matrix()
    {
        hipFree(data);
    }

    // move constructor
    Matrix(Matrix&& other) noexcept : rows(other.rows), cols(other.cols), data(other.data)
    {
        other.data = nullptr;
    }

    // move assignment operator
    Matrix& operator=(Matrix&& other) noexcept
    {
        if (this != &other)
        {
            hipFree(data);

            rows = other.rows;
            cols = other.cols;
            data = other.data;

            other.data = nullptr;
        }
        return *this;
    }

    // copy constructor
    Matrix(const Matrix& other) : rows(other.rows), cols(other.cols)
    {
        hipMalloc(&data, rows * cols * sizeof(float));
        hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
    }

    // copy assignment operator
    Matrix& operator=(const Matrix& other)
    {
        if (this != &other)
        {
            hipFree(data);

            rows = other.rows;
            cols = other.cols;

            hipMalloc(&data, rows * cols * sizeof(float));
            hipMemcpy(data, other.data, rows * cols * sizeof(float), hipMemcpyDeviceToDevice);
        }
        return *this;
    }

    void print()
    {
        float* dup = (float*)malloc(rows * cols * sizeof(float));
        hipMemcpy(dup, data, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "////////////////////////////////////////\n";

        for (int i = 0; i < rows; i++)
        {
            for (int j = 0; j < cols; j++)
            {
                std::cout << dup[(i * cols) + j] << "  ";
            }
            std::cout << "\n";
        }

        std::cout << "////////////////////////////////////////\n";

        free(dup);
    }

    Matrix operator+(const Matrix& other) const
    {
        if (this->cols != other.cols || this->rows != other.rows)
        {
            throw std::invalid_argument("Matrix dimensions do not match for addition.");
        }

        Matrix result(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));
        
        matrixAdd <<< 1, this->rows * this->cols >>> (this->data, other.data, result.data); // TODO make more sophisticated for the love of god
        hipDeviceSynchronize();

        return result;
    }

    Matrix operator-(const Matrix& other) const
    {
        if (this->cols != other.cols || this->rows != other.rows)
        {
            throw std::invalid_argument("Matrix dimensions do not match for subtraction.");
        }

        Matrix result(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));

        matrixSub <<< 1, this->rows * this->cols >>> (this->data, other.data, result.data); // TODO make more sophisticated for the love of god
        hipDeviceSynchronize();

        return result;
    }

    Matrix& operator+=(const Matrix& other)
    {
        if (this->cols != other.cols || this->rows != other.rows)
        {
            throw std::invalid_argument("Matrix dimensions do not match for addition.");
        }
        matrixAdd << < 1, this->rows* this->cols >> > (this->data, other.data, this->data); // calling regular add kernel but w/ left matrix as result
        hipDeviceSynchronize();

        return *this;
    }

    Matrix& operator-=(const Matrix& other)
    {
        if (this->cols != other.cols || this->rows != other.rows)
        {
            throw std::invalid_argument("Matrix dimensions do not match for addition.");
        }
        matrixSub << < 1, this->rows* this->cols >> > (this->data, other.data, this->data); // calling regular add kernel but w/ left matrix as result
        hipDeviceSynchronize();

        return *this;
    }

    Matrix operator*(const Matrix& other) const
    {
        if (this->cols != other.rows)
        {
            throw std::invalid_argument("Matrix dimensions do not match for multiplication.");
        }

        Matrix result(this->rows, other.cols, 0.0f);
        hipMalloc(&result.data, this->rows * other.cols * sizeof(float));

        matrixDot <<<1, this->rows * other.cols>>> (this->data, other.data, result.data, this->cols, other.cols);
        hipDeviceSynchronize();
        
        return result;
    }

    // Matrix scalar multiplication!!
    Matrix operator*(const float scalar) const
    {
        Matrix result(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));

        matrixScalarMult <<< 1, this->rows * this->cols >>> (this->data, scalar, result.data);
        hipDeviceSynchronize();

        return result;
    }

    Matrix& operator*=(const float scalar)
    {
        matrixScalarMult <<< 1, this->rows* this->cols >>> (this->data, scalar, this->data);
        hipDeviceSynchronize();

        return *this;
    }

    Matrix operator/(const float scalar) const
    {
        Matrix result(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));

        matrixScalarDiv <<< 1, this->rows* this->cols >>> (this->data, scalar, result.data);
        hipDeviceSynchronize();

        return result;
    }

    Matrix& operator/=(const float scalar)
    {
        matrixScalarDiv << < 1, this->rows* this->cols >> > (this->data, scalar, this->data);
        hipDeviceSynchronize();

        return *this;
    }

    Matrix T() const
    {
        Matrix transposed = Matrix(this->cols, this->rows);
        hipMalloc(&transposed.data, this->cols * this->rows * sizeof(float));

        matrixTranspose <<< 1, this->rows * this->cols >>> (this->data, transposed.data, this->rows, this->cols);
        hipDeviceSynchronize();
        
        return transposed;
    }

    Matrix relu() const
    {
        Matrix result = Matrix(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));

        matrixRELU <<< 1, this->rows * this->cols >>> (this->data, result.data);
        hipDeviceSynchronize();

        return result;
    }

    Matrix exp() const
    {
        Matrix result = Matrix(this->rows, this->cols);
        hipMalloc(&result.data, this->rows * this->cols * sizeof(float));

        matrixExp << < 1, this->rows* this->cols >> > (this->data, result.data);
        hipDeviceSynchronize();

        return result;
    }

    // operator overload for float addition, subtraction, multiplication, (division?? integer division? modulo???) on matrix, hadamard (mat1.had(mat2);)
    // or had(mat1, mat2);
    
};

// global non-member function for making matrix-scalar multiplication commutative
Matrix operator*(const float scalar, const Matrix mat)
{
    return mat * scalar;
}

Matrix operator/(const float scalar, const Matrix mat) // TODO test
{
    Matrix result(mat.rows, mat.cols);
    hipMalloc(&result.data, mat.rows * mat.cols * sizeof(float));

    matrixScalarReciprocal <<< 1, mat.rows * mat.cols >>> (scalar, mat.data, result.data);
    hipDeviceSynchronize();
    return result;
}

// hadamard product (element-wise matrix multiplication)
Matrix had(const Matrix mat1, const Matrix mat2)
{
    if (mat1.cols != mat2.cols || mat1.rows != mat2.rows)
    {
        throw std::invalid_argument("Matrix dimensions do not match for hadamard product.");
    }
    Matrix result = Matrix(mat1.rows, mat1.cols);
    hipMalloc(&result.data, mat1.rows * mat1.cols * sizeof(float));

    matrixHad << < 1, mat1.rows * mat1.cols >> > (mat1.data, mat2.data, result.data);
    hipDeviceSynchronize();

    return result;
}

Matrix avgToColumn(const Matrix mat)
{
    Matrix result = Matrix(mat.rows, 1, 0.0f);
    hipMalloc(&result.data, mat.rows * sizeof(float));

    avgToColumn <<< 1, mat.rows >>> (mat.data, result.data, mat.cols);
    hipDeviceSynchronize();

    return result;
}

// TODO applying (atomic) math functions, saving matrices, loading matrices, more inplace functions (for example, distinguish using f(), f_() maybe)
// probably not necessary given that we already have move semantics in place
// long term: literally make AUTODIFF for backpropagation, infrastructure for mini-batch inference/gradient descent, other stuff necessary for
// moderately-fledged NN library, make OOP stuff more encapsulated
// 

// i can do stack arrays of whatever dimension just fine, just need to pass pointer to array into the constructor
// heap arrays: should be allocated as 1d. i can make that happen with the dataloader helper functions. can't think of any other instance where that would
// actually be used.

// already did: exp, relu
// list of math functions to implement (sensible derivatives): pow, root, exp, log (just base e), sin, cos, relu, loss function derivatives
// matrix norms (need to take derivatives of these too!)
// hadamard matrix multiplication
// sums, axis sums
// data loaders, whatever would be convenient as an interface for the dataloading functions in the neural network class

// add more boundary checking, imporant once you make block/thread deployment more involved

int main()
{
    /*
    std::cout << "testing out matrix creation\n";
    Matrix a = Matrix(5, 5, Matrix::InitType::Random);
    a.print();
    Matrix b = Matrix(5, 5, 3);
    b.print();
    Matrix c = Matrix(5, 5, Matrix::InitType::Identity);
    c.print();
    std::cout << "testing out matrix addition/subtraction\n";
    Matrix d = a + b + c;
    d.print();

    Matrix e = a - c;
    e.print();
    std::cout << "testing out matrix multiplication\n";
    Matrix f = Matrix(2, 2, 5);
    Matrix g = Matrix(2, 3, 5);
    Matrix h = f * g;

    f.print();
    g.print();
    h.print();

    Matrix i = Matrix(4, 4, Matrix::InitType::Identity);
    Matrix j = Matrix(4, 4, Matrix::InitType::Random);
    Matrix k = i * j;

    i.print();
    j.print();
    k.print();

    Matrix l = Matrix(10, 10, Matrix::InitType::Random);
    Matrix m = Matrix(10, 1, 0.1);
    Matrix n = l * m;

    l.print();
    m.print();
    n.print();
    
    float stackMatrix[4][3] = { {1,2,3},{4,5,6},{7,8,9},{10,11,12} };
    float* heapMatrix = new float[9] {4,2,3,-8,2.5,6,1,0,1};
    Matrix mat1 = Matrix(4, 3, *stackMatrix);
    Matrix mat2 = Matrix(3, 3, heapMatrix);
    mat1.print();
    mat2.print();
    Matrix mat3 = mat1 * mat2;
    mat3.print();
    
    std::cout << "testing transposes:\n";
    float thearr[5][4] = { {1,2,3,4},{1,0,0,0},{1,1,0,0},{1,1,1,0},{1,1,1,1} };
    Matrix transposeTest = Matrix(5, 4, *thearr);
    transposeTest.print();
    Matrix max = transposeTest.T();
    max.print();

    k = Matrix(4, 4, 5);
    i.print();
    j.print();
    k.print();
    std::cout << "testing operator overloading for adding/subtracting in place\n";

    i += j;
    i.print();
    i -= j + k;
    i.print();

    Matrix first = Matrix(5, 5, Matrix::InitType::Identity);
    Matrix second = Matrix(5, 3, Matrix::InitType::He);
    first.print();
    second.print();
    Matrix result1 = first * 5;
    Matrix result2 = 6 * first * second * 5;
    result1.print();
    result2.print();

    Matrix jeff = Matrix(5, 5, Matrix::InitType::Identity);
    jeff.print();
    jeff *= 72;
    jeff.print();

    Matrix joe = Matrix(4, 4, 6);
    joe.print();
    Matrix mama = joe / 2;
    mama.print();
    joe /= 4;
    joe.print();
    */

    std::cout << "testing RELU\n";
    float thearray[5][4] = { {1,-3.2359875,3,4},{1,-5,-3,0},{1,1,-157,0},{1,1,1,-1},{1,-1,1,1} };
    Matrix reluTest = Matrix(5, 4, *thearray);
    reluTest.print();
    Matrix reluTested = reluTest.relu();
    reluTested.print();

    std::cout << "testing exp\n";
    float exparray[5][4] = { {1, -3.2359875, 3, 4}, { 12,-5,-3,0 }, { 5,6,-157,7 }, { 9,8,1,-1 }, { 15.8,-1,1,1 } };
    Matrix expTest = Matrix(5, 4, *exparray);
    expTest.print();
    Matrix expTested = expTest.exp();
    expTested.print();

    std::cout << "testing hadamard product\n";
    float matOnearr[4][4] = { {3, 5, -9, 6}, {56, 43, 2, 5}, {-5, -14, -.05403, 0}, {.001, 25, 26, 1} };
    float matTwoarr[4][4] = { {4, 2, 22, 1}, {2, 3, 4, 5}, {-4, 4, 2, -.5}, {4, 0, 0, 4} };
    Matrix matOne = Matrix(4, 4, *matOnearr);
    Matrix matTwo = Matrix(4, 4, *matTwoarr);
    matOne.print();
    matTwo.print();
    Matrix matThree = had(matOne, matTwo);
    matThree.print();
}

/*

I think that i need to add reciprocal function and use that in scalar division operation

THE MINIBATCH WILL JUST BE A MATRIX OH MY GOD
WE LITERALLY JUST HAVE THE POINTER THROUGH IT INDEXED WITH A FOR LOOP 

from there we can do whatever averaging we need to do over the vectors for gradient descent.

all of this batch training loop stuff is for later though

i will have to make NN class first.
functions to create various layers/activations

whoa there buddy scope creep
*/